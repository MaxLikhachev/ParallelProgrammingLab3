#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cmath>

using namespace std;

#define BLOCK_SIZE 8
#define N 32

const float degreesToRadiansCoefficient = 0.0174533;
const int minValue = 0;
const int maxValue = 360;

/* TODO:
* fix shared memory
* try to fix constant memory
*/

hipError_t calculateWithCuda(float*c, float*a, float*b, unsigned int size);

__global__ void globalCalculateKernel(float*c, float*a, float*b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i * j] = sin(a[i * j]) * sin(a[i * j]) + cos(b[i * j]) * cos(b[i * j]) * cos(b[i * j]);
}

__global__ void sharedCalculateKernel(float* c, float* a, float* b, unsigned int size)
{
    __shared__ float shared_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_b[BLOCK_SIZE][BLOCK_SIZE];
    float shared_c_temp= 0;

    for (int i = size * BLOCK_SIZE * blockIdx.y; i <= size * BLOCK_SIZE * blockIdx.y + size - 1; i += BLOCK_SIZE)
    {
        shared_a[threadIdx.x][threadIdx.y] = a[i + size * threadIdx.y + threadIdx.x];
        shared_b[threadIdx.x][threadIdx.y] = b[i + size * threadIdx.y + threadIdx.x];
        shared_c_temp = shared_a[threadIdx.x][threadIdx.y] * shared_a[threadIdx.x][threadIdx.y];
    }
    c[size * BLOCK_SIZE * blockIdx.y + size * BLOCK_SIZE * blockIdx.y + threadIdx.y * size + threadIdx.x] = shared_c_temp;
}
/*
__constant__ float constant_a[N * N];
__constant__ float constant_b[N * N];
__global__ void constantCalculateKernel(float* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i * j] = sin(constant_a[i * j]) * sin(constant_a[i * j]) + cos(constant_b[i * j]) * cos(constant_b[i * j]) * cos(constant_b[i * j]);
}
*/
bool isCalculationCorrect(int arraySize, float* c, const float* a, const float* b)
{
    bool isError = true;
    for (int i = 0; i < arraySize && isError; i++)
        for (int j = 0; j < arraySize && isError; j++)
            isError = c[i * j] != sin(a[i * j]) * sin(a[i * j]) + cos(b[i * j]) * cos(b[i * j]) * cos(b[i * j]);
    return isError;
}

void initRandom(int arraySize, float* a) 
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = minValue + rand() % maxValue * degreesToRadiansCoefficient;

}

void initNull(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = 0;
}

void display(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
    {
        for (int j = 0; j < arraySize; j++)
            cout << a[i * arraySize + j] << " ";
        cout << endl;
    }
}

int main()
{
    srand(time(NULL));

    cout << "Enter array size: ";
    int arraySize = 0;
    cin >> arraySize;

    float* a = new float[arraySize * arraySize];
    float* b = new float[arraySize * arraySize];
    float* c = new float[arraySize * arraySize];

    initRandom(arraySize, a);
    initRandom(arraySize, b);
    initNull(arraySize, c);

    // Add matrixes in parallel.
    hipError_t cudaStatus = calculateWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        cout << "calculateWithCuda failed!\n";
       return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        cout << "hipDeviceReset failed!\n";
        return 1;
    }

    return 0;
}


// Helper function for using CUDA to add matrixes in parallel.
hipError_t calculateWithCuda(float*c, float*a, float*b, unsigned int size)
{
    float* dev_a;
    float* dev_b;
    float* dev_c;

    hipError_t cudaStatus;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float KernelTime;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three matrixes (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input matrixes from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.

    int numBlocks = BLOCK_SIZE;
    dim3 threadsPerBlock(N, N);

    cout << "Config settings: arraySize = " << size << ", numBlocks = " << numBlocks << ", threadsPerBlock(" << N << ", " << N << ")\n";


    // Global memory
    hipEventRecord(start, 0);
    globalCalculateKernel <<<numBlocks, threadsPerBlock>>> (dev_c, dev_a, dev_b);
    // if (!isCalculationCorrect(size, dev_c, dev_a, dev_b)) cout << "Calculation Error\n";
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    cout << "\nGlobal result: " << KernelTime <<  " milliseconds\n";

    // Shared memory
    hipEventRecord(start, 0);
    sharedCalculateKernel << <numBlocks, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);
    // if (!isCalculationCorrect(size, dev_c, dev_a, dev_b)) cout << "Calculation Error\n";
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    cout << "\Shared result: " << KernelTime << " milliseconds\n";

    // Constant memory
    /*
    hipEventRecord(start, 0);
    float constant_a[N * N];
    float constant_b[N * N];
    cudaStatus = hipMemcpy(constant_a, a, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(constant_b, b, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    constantCalculateKernel << <numBlocks, threadsPerBlock >> > (dev_c);
    if (!isCalculationCorrect(size, dev_c, dev_a, dev_b)) cout << "Calculation Error\n";
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    cout << "\nConstant result: " << KernelTime << " milliseconds\n";
*/
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "globalCalculateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching globalCalculateKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output matrix from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, (N * N) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
