#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

const float degreesToRadiansCoefficient = 0.0174533;
const int minValue = 0;
const int maxValue = 360;

hipError_t addWithCuda(float*c, float*a, float*b, unsigned int size);

__global__ void addKernel(float*c, const float*a, const float*b)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    c[i * j] = a[i * j] + b[i * j];
}

void initRandom(int arraySize, float* a) 
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = minValue + rand() % maxValue * degreesToRadiansCoefficient;

}

void initNull(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = 0;
}

void display(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
    {
        for (int j = 0; j < arraySize; j++)
            cout << a[i * arraySize + j] << " ";
        cout << endl;
    }
}

int main()
{
    srand(time(NULL));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    cout << "Enter array size: ";
    int arraySize = 0;
    cin >> arraySize;
    cout <<  " Array size: " << arraySize << endl;
    //const float a[arraySize][arraySize] = {{ 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }};
    //const float b[arraySize][arraySize] = { { 10, 20, 30, 40, 50 }, { 10, 20, 30, 40, 50 },{ 10, 20, 30, 40, 50 },{ 10, 20, 30, 40, 50 },{ 10, 20, 30, 40, 50 }, };
    //float c[arraySize][arraySize] = { {0} };

    float* a = new float[arraySize * arraySize];
    float* b = new float[arraySize * arraySize];
    float* c = new float[arraySize * arraySize];

    initRandom(arraySize, a);
    initRandom(arraySize, b);
    initNull(arraySize, c);

    // cout << "A\n";
    // display(arraySize, a);
    // cout << "B\n";
    // display(arraySize, b);
    // cout << "C\n";
    // display(arraySize, c);

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        cout << "addWithCuda failed!\n";
       return 1;
    }

    // cout << c[0][0] << c[0][1] << c[0][2] << c[0][3] << c[0][4];
    display(arraySize, c);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        cout << "hipDeviceReset failed!\n";
        return 1;
    }

    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float*c, float*a, float*b, unsigned int size)
{
    float* dev_a;
    float* dev_b;
    float* dev_c;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.

    int numBlocks = 1;
    dim3 threadsPerBlock(size, size);
    addKernel <<<numBlocks, threadsPerBlock>>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, (size * size) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
