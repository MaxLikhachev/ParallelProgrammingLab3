#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cmath>

using namespace std;

#define BLOCK_SIZE 8
#define N 32

const float degreesToRadiansCoefficient = 0.0174533;
const int minValue = 0;
const int maxValue = 360;

hipError_t addWithCuda(float*c, float*a, float*b, unsigned int size);

__global__ void addKernel(float*c, const float*a, const float*b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i * j] = sin(a[i * j]) * sin(a[i * j]) + cos(b[i * j]) * cos(b[i * j]) * cos(b[i * j]);
}

bool isCalculationCorrect(int arraySize, float* c, const float* a, const float* b)
{
    bool isError = true;
    for (int i = 0; i < arraySize && isError; i++)
        for (int j = 0; j < arraySize && isError; j++)
            isError = c[i * j] != sin(a[i * j]) * sin(a[i * j]) + cos(b[i * j]) * cos(b[i * j]) * cos(b[i * j]);
    return isError;
}

void initRandom(int arraySize, float* a) 
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = minValue + rand() % maxValue * degreesToRadiansCoefficient;

}

void initNull(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = 0;
}

void display(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
    {
        for (int j = 0; j < arraySize; j++)
            cout << a[i * arraySize + j] << " ";
        cout << endl;
    }
}

int main()
{
    srand(time(NULL));

    cout << "Enter array size: ";
    int arraySize = 0;
    cin >> arraySize;
    cout <<  "Array size: " << arraySize << endl;
    //const float a[arraySize][arraySize] = {{ 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }, { 1, 2, 3, 4, 5 }};
    //const float b[arraySize][arraySize] = { { 10, 20, 30, 40, 50 }, { 10, 20, 30, 40, 50 },{ 10, 20, 30, 40, 50 },{ 10, 20, 30, 40, 50 },{ 10, 20, 30, 40, 50 }, };
    //float c[arraySize][arraySize] = { {0} };

    float* a = new float[arraySize * arraySize];
    float* b = new float[arraySize * arraySize];
    float* c = new float[arraySize * arraySize];

    initRandom(arraySize, a);
    initRandom(arraySize, b);
    initNull(arraySize, c);

    // cout << "A\n";
    // display(arraySize, a);
    // cout << "B\n";
    // display(arraySize, b);
    // cout << "C\n";
    // display(arraySize, c);

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        cout << "addWithCuda failed!\n";
       return 1;
    }

    // cout << c[0][0] << c[0][1] << c[0][2] << c[0][3] << c[0][4];
    // display(arraySize, c);
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        cout << "hipDeviceReset failed!\n";
        return 1;
    }

    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float*c, float*a, float*b, unsigned int size)
{
    float* dev_a;
    float* dev_b;
    float* dev_c;

    hipError_t cudaStatus;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float KernelTime;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.

    int numBlocks = BLOCK_SIZE;
    dim3 threadsPerBlock(N, N);

    hipEventRecord(start, 0);
    cout << "Config: numBlocks = " << numBlocks << ", threadsPerBlock(" << N << ", " << N << ")" << endl;
    addKernel <<<numBlocks, threadsPerBlock>>> (dev_c, dev_a, dev_b);
    if (!isCalculationCorrect(size, a, b, c)) cout << "Calculation Error\n";
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    cout << "\nGlobal result: " << KernelTime <<  " milliseconds\n";
    //printf("KernelTime: %.2f milliseconds\n", KernelTime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, (N * N) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
